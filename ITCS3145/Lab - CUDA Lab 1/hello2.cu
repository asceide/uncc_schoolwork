#include <stdio.h>
#include <hip/hip_runtime.h>
// __global__ keyword specifies a device kernel function
__global__ void cuda_hello() {

   printf("Hello from GPU blockIdx.x=%d blockIdx.y=%d blockIdx.z=%d threadIdx.x=%d threadIdx.y=%d threadIdx.z=%d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}

 

int main() {
    printf("Hello World from CPU!\n");
    // Call a device function from the host: a kernel launch Which will print from the device
    dim3 grid(16,16);
    dim3 block(16,16);
    cuda_hello<<<grid, block>>>();

     // This call waits for all of the submitted GPU work to complete

     hipDeviceSynchronize();

      return 0;

}
